
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void read_mem(char* state){
      *state = threadIdx.x + blockIdx.x * blockDim.x + 0xffffffff;
      //for(int i=0; i<1; i++){
      //printf("%x\n", *((char *)0x6048a0000));
      //printf("%x\n", *((char *)0x6048a1000));
      printf("%p\n", state);
      if((long)state == 0x6048a0000){
	printf("Probe : Before allocated\n");
        unsigned long long start_address = 0x6048a0000, curr_address = 0x0;
        int count = 0;
	printf("Probe : 0MB to 64MB before allocated\n");
	while(count < 5){
		curr_address = start_address - count*0x1000000;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : 64MB to 72MB before allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 9){
		curr_address = start_address - count*0x100000;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : 72MB to 73MB before allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 16){
		curr_address = start_address - count*0x10000;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	curr_address = curr_address - 0x1;
	printf("%p => %x\n", curr_address, *((char *)curr_address));
      }
      if((long)state == 0x8cd4a0000){
	printf("Probe : Allocated\n");
        unsigned long long start_address = 0x8cd4a0000, curr_address = 0x0;
        int count = 0;
	while(count < 10){
		curr_address = start_address + count*0x100000;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : Last 1MB of allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 16){
		curr_address = start_address + count*0x10000;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : Last 64KB of allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 16){
		curr_address = start_address + count*0x1000;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : Last 4KB of allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 16){
		curr_address = start_address + count*0x100;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : Last 512B of allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 16){
		curr_address = start_address + count*0x10;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
	printf("Probe : Last 32B of allocated\n");
        count = 0;
	start_address = curr_address;
	while(count < 16){
		curr_address = start_address + count*0x1;
		count++;
		printf("%p => %x\n", curr_address, *((char *)curr_address));
	}
      }
      //printf("[%p] = %x\n", 0x0, *((char *)0x0));
      //printf("[%p] = %x\n", 0x1, *((char *)0x1));
      //*state = *((char *)i);
      //}
      //printf("state from GPU = %x (at %p)\n", *state, state);
}

int main(){
    char *h_state, *d_state;
    h_state = (char *)malloc(sizeof(int));
    hipError_t cerr;
    unsigned long long alloc_size = 1024*1024*1024;
    unsigned long long tot_alloc_size = 0;
    for(int i=0; i<11; i++){
	    cerr = hipMalloc(&d_state, sizeof(char)*alloc_size);
	    if(cerr != hipSuccess){
		    printf("hipMalloc failed : %s\n", hipGetErrorString(cerr));
	    }else{
		    tot_alloc_size+=alloc_size;
		    printf("Allocated another 1GB...total allocated = %llu\n", tot_alloc_size/(1024*1024*1024));
		    read_mem<<<1,1>>>(d_state);
		    hipDeviceSynchronize();
	    }
    }
    alloc_size = 1024*1024*100; tot_alloc_size = 0;
    for(int i=0; i<1; i++){
	    cerr = hipMalloc(&d_state, sizeof(char)*alloc_size);
	    if(cerr != hipSuccess){
		    printf("hipMalloc failed : %s\n", hipGetErrorString(cerr));
	    }else{
		    tot_alloc_size+=alloc_size;
		    printf("Allocated another 100 MB...total allocated = %llu\n", tot_alloc_size/(1024*1024*100));
		    read_mem<<<1,1>>>(d_state);
		    hipDeviceSynchronize();
	    }
    }
    alloc_size = 1024*1024*10; tot_alloc_size = 0;
    for(int i=0; i<5; i++){
	    cerr = hipMalloc(&d_state, sizeof(char)*alloc_size);
	    if(cerr != hipSuccess){
		    printf("hipMalloc failed : %s\n", hipGetErrorString(cerr));
	    }else{
		    tot_alloc_size+=alloc_size;
		    printf("Allocated another 10 MB...total allocated = %llu\n", tot_alloc_size/(1024*1024*10));
		    read_mem<<<1,1>>>(d_state);
		    cerr = hipDeviceSynchronize();
		    if(cerr != hipSuccess){
			    printf("hipDeviceSynchronize failed : %s\n", hipGetErrorString(cerr));
		    }
	    }
    }
#if 0
    long long address = 0xF00000000;
    unsigned count = 0;
    while(count++ < 1000){
    read_mem<<<1,1>>>(d_state, address);
    address = address + 0x100000;
    }
    cudaMemcpy(h_state, d_state, sizeof(int), cudaMemcpyDeviceToHost);
    printf("state from CPU = %x\n", *h_state);
#endif
    return 0;
}
